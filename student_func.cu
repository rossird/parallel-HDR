#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__device__ float min_float(float a, float b) {
    return a < b ? a : b;
}

//Calculate a min scan
__global__
void min_scan(float* array, int size, float* min) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    for (int offset = 1; offset < size; offset *= 2) {
        if (i >= offset) {
            array[i] = min_float(array[i - offset], array[i]);
        }
    }
    
    //*min = array[size-1];

}

//Calculate a max scan
__global__
void max_scan(const float* array, size_t size, float* max) {

}

//Calculate a histogram
__global__
void histogram(const float* const lum, float lumMin, float lumRange, size_t numBins, int* histogram) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int bin = (lum[i] - lumMin) / lumRange * numBins;
    atomicAdd(&histogram[bin], 1);
}

//Calculate an exclusive scan
__global__
void exclusive_scan(float* input, size_t size, unsigned int* const cdf) {

}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
    
    int* d_histogram;
    float* d_max;
    float* d_min;
    float* d_temp;
    float range;
    
    //Allocate memory
    checkCudaErrors(hipMalloc(&d_max, sizeof(float)));
    checkCudaErrors(hipMalloc(&d_min, sizeof(float)));
    checkCudaErrors(hipMalloc(&d_histogram, sizeof(int) * numBins));
    checkCudaErrors(hipMalloc(&d_temp, sizeof(float) * numCols * numRows));
    
    //Initialize memory
    checkCudaErrors(hipMemset(d_histogram, 0, sizeof(int) * numBins));
    checkCudaErrors(hipMemcpy(d_temp, d_logLuminance, sizeof(float) * numCols * numRows, hipMemcpyDeviceToDevice));
    
    //Min scan to find min_logSum
    min_scan<<<numRows, numCols>>>(d_temp, numCols * numRows, d_min);
    //checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));
    
    //Max scan to find max_logSum
    //max_scan<<<numCols, numRows>>>(d_logLuminance, numCols * numRows, d_max);
    //checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));
    
    //Find range
    range = max_logLum - min_logLum;
    
    //Generate histogram
    //histogram<<<numCols, numRows>>>(d_logLuminance, min_logLum, range, numBins, d_histogram);
    
    //Exclusive scan to find cdf
    //exclusive_scan<<<numCols, numRows>>>(d_logLuminance, numCols * numRows,  d_cdf);
    
    //free(d_max);
    //free(d_min);
    //free(d_histogram);
}
